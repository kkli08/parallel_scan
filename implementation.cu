#include "hip/hip_runtime.h"
#include "implementation.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

void printSubmissionInfo()
{
    char nick_name[] = "Kenji-Fujima";
    char student_first_name[] = "Damian";
    char student_last_name[] = "Li";
    char student_student_number[] = "1005842554";

    printf("*******************************************************************************************************\n");
    printf("Submission Information:\n");
    printf("\tnick_name: %s\n", nick_name);
    printf("\tstudent_first_name: %s\n", student_first_name);
    printf("\tstudent_last_name: %s\n", student_last_name);
    printf("\tstudent_student_number: %s\n", student_student_number);
}

#define MAX_THREADS_PER_BLOCK 1024

// Kernel 1: Per-block inclusive scan
__global__ void block_inclusive_scan_kernel(const int32_t *d_input, int32_t *d_output, int32_t *d_block_sums, size_t n)
{
    extern __shared__ int32_t s_data[];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;

    // Load data into shared memory
    s_data[tid] = (gid < n) ? d_input[gid] : 0;
    __syncthreads();

    // Inclusive scan within the block using Kogge-Stone algorithm
    for (int offset = 1; offset < blockDim.x; offset <<= 1)
    {
        int temp = 0;
        if (tid >= offset)
            temp = s_data[tid - offset];
        __syncthreads();
        s_data[tid] += temp;
        __syncthreads();
    }

    // Write the scanned data to global memory
    if (gid < n)
    {
        d_output[gid] = s_data[tid];
    }

    // Write the total sum of this block to d_block_sums
    if (tid == blockDim.x - 1)
    {
        d_block_sums[blockIdx.x] = s_data[tid];
    }
}

// CPU function to perform prefix scan
void cpu_prefix_scan(const int32_t *input, int32_t *output, size_t n)
{
    output[0] = input[0];
    for (size_t i = 1; i < n; i++)
    {
        output[i] = output[i - 1] + input[i];
    }
}

// Kernel 2: Adjust scanned data with block sums
__global__ void adjust_with_block_sums_kernel(int32_t *d_output, const int32_t *d_scanned_block_sums, size_t n)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid < n && blockIdx.x > 0)
    {
        d_output[gid] += d_scanned_block_sums[blockIdx.x - 1];
    }
}

// Main inclusive scan implementation
void implementation(const int32_t *d_input, int32_t *d_output, size_t size)
{
    int threadsPerBlock = 512;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate memory for block sums
    int32_t *d_block_sums;
    hipMalloc(&d_block_sums, numBlocks * sizeof(int32_t));

    // Kernel 1: Per-block inclusive scan
    size_t sharedMemSize = threadsPerBlock * sizeof(int32_t);
    block_inclusive_scan_kernel<<<numBlocks, threadsPerBlock, sharedMemSize>>>(d_input, d_output, d_block_sums, size);
    hipDeviceSynchronize();

    // Copy block sums to host memory
    int32_t *h_block_sums = (int32_t *)malloc(numBlocks * sizeof(int32_t));
    hipMemcpy(h_block_sums, d_block_sums, numBlocks * sizeof(int32_t), hipMemcpyDeviceToHost);

    // CPU prefix scan on block sums
    int32_t *h_scanned_block_sums = (int32_t *)malloc(numBlocks * sizeof(int32_t));
    cpu_prefix_scan(h_block_sums, h_scanned_block_sums, numBlocks);

    // Copy scanned block sums back to device memory
    int32_t *d_scanned_block_sums;
    hipMalloc(&d_scanned_block_sums, numBlocks * sizeof(int32_t));
    hipMemcpy(d_scanned_block_sums, h_scanned_block_sums, numBlocks * sizeof(int32_t), hipMemcpyHostToDevice);

    // Kernel 2: Adjust with scanned block sums
    adjust_with_block_sums_kernel<<<numBlocks, threadsPerBlock>>>(d_output, d_scanned_block_sums, size);
    hipDeviceSynchronize();

    // Free allocated memory
    free(h_block_sums);
    free(h_scanned_block_sums);
    hipFree(d_block_sums);
    hipFree(d_scanned_block_sums);
}
